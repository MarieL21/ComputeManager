#include "ComputeManager.h"

#if (CUDART_VERSION <= 6050)
int hipInvalidDeviceId = -1;
#endif

int GetSPcores(const hipDeviceProp_t& devProp);

int IsValidDevice(int device_id)
{
	return device_id != hipInvalidDeviceId;	
}

ComputeManager::ComputeManager()
:device_id(hipInvalidDeviceId),
 sm_count(2),
 is_initialized(true)	
{	
}

hipError_t ComputeManager::GetComputeDevices(std::vector<ComputeDeviceLib> &devices)
{
	int device_count = 0;
	CHECK_CUDA(hipGetDeviceCount(&device_count));
	if (!device_count)
	{
		log_err << "No CUDA enabled devices found." << log_endl;
		return hipSuccess;		
	}
	devices.resize(device_count);
	hipDeviceProp_t prop;	
	for (int i = 0; i < device_count; ++i)
	{
		devices[i].deviceId = i;
		CHECK_CUDA(hipGetDeviceProperties(&prop, i));		
		devices[i].deviceName = std::string(prop.name);
	}	
	return hipSuccess;	
}

bool ComputeManager::CheckNoErrIsSameAsApi(int app_success)
{
	return app_success == hipSuccess;	
}

hipError_t ComputeManager::Release()
{
	device_id = hipInvalidDeviceId;	
	return hipSuccess;	
}

hipError_t ComputeManager::SetComputeDevice(ComputeDeviceLib deviceToUse)
{
	if (IsValidDevice(deviceToUse.deviceId))
	{
		CHECK_CUDA(hipSetDevice(deviceToUse.deviceId));
		device_id = deviceToUse.deviceId;		
		return hipSuccess;
	}
	else
	{
		log_err << "Invalid deviceId." << log_endl;		
	}
	return hipErrorInvalidValue;	
}

hipError_t ComputeManager::Initialize() const
{
	if (IsValidDevice(device_id))
	{
		log_inf << "CUDA device has already been configured!" << log_endl;
		return hipSuccess;		
	}
	CHECK_CUDA(ChooseBestDeviceAvailable(device_id));
	if (IsValidDevice(device_id))
	{
		hipDeviceProp_t prop;
		CHECK_CUDA(hipGetDeviceProperties(&prop, device_id));
		log_inf << "Using CUDA device: " << prop.name << " [" << device_id <<  "]"
			   ", compute capability: " << prop.major << "." << prop.minor
				<< log_endl;
		sm_count = prop.multiProcessorCount;
		CHECK_CUDA(hipSetDevice(device_id));		
	}
	is_initialized = true;
	return hipSuccess;
}

bool ComputeManager::IsInitialized() const
{
	return is_initialized;	
}

int ComputeManager::GetSmCount() const
{
	return sm_count;	
}

hipError_t ComputeManager::ChooseBestDeviceAvailable(int& device_id) const
{
	std::vector<ComputeDeviceLib> devices;
	CHECK_CUDA(GetComputeDevices(devices));

	hipDeviceProp_t prop;
	float max_device_rating = 0;
	int best_device = hipInvalidDeviceId;	
	for (int i = 0; i < (int)devices.size(); ++i)
	{		
		CHECK_CUDA(hipGetDeviceProperties(&prop, i));
		float curr_device_rating = 2.0 * prop.clockRate * GetSPcores(prop)/1e6;
		if (curr_device_rating > max_device_rating)
		{
			max_device_rating = curr_device_rating;
			best_device = i;			
		}
		log_inf << "device: " << devices[i].deviceName << " "
				<< curr_device_rating << " GFLOPS" << log_endl;
	}
	device_id = best_device;
	return hipSuccess;	
}

int GetSPcores(const hipDeviceProp_t& devProp)
{  
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major)
	{
		case 2: // Fermi
			if (devProp.minor == 1) cores = mp * 48;
			else cores = mp * 32;
			break;
		case 3: // Kepler
			cores = mp * 192;
			break;
		case 5: // Maxwell
			cores = mp * 128;
			break;
		case 6: // Pascal
			if (devProp.minor >= 1) cores = mp * 128;
			else if (devProp.minor == 0) cores = mp * 64;
			else log_inf << "Unknown device type!" << log_endl; 
			break;
		case 7: // Volta
			if (devProp.minor == 0) cores = mp * 64;
			else log_inf << "Unknown device type!" << log_endl;
			break;
		default:
			log_inf << "Unknown device type!" << log_endl; 
			break;
	}
    return cores;
}
